#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <chrono>
using namespace std;

__global__ void add(int *a, int *b)
{
    for(int i = 0; i < 1000000; i++)
        a[i] += b[i];
}
void addCPU(int *a, int *b)
{
    for(int i = 0; i < 1000000; i++)
        a[i] += b[i];
}
int main()
{
    std::chrono::_V2::system_clock::time_point checkpoint[10];
    int n = 1000000;
    int *a = (int *)malloc(n * sizeof(int));
    int *b = (int *)malloc(n * sizeof(int));
    int *da, *db;

    srand(time(NULL));
    for(int i = 0; i < n; i++)
    {
        a[i] = rand() % 152;
        b[i] = rand() % 314;
    }

    hipMalloc(&da, sizeof(int) * n);
    hipMalloc(&db, sizeof(int) * n);
    hipMemcpy(da, a, sizeof(int) * n, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(int) * n, hipMemcpyHostToDevice);

    checkpoint[0] = std::chrono::high_resolution_clock::now();
    add<<<1,1>>>(da, db);
    hipDeviceSynchronize();
    checkpoint[1] = std::chrono::high_resolution_clock::now();
    addCPU(a, b);
    checkpoint[2] = std::chrono::high_resolution_clock::now();

    auto x = std::chrono::duration_cast<std::chrono::microseconds>(checkpoint[1] - checkpoint[0]).count();
    auto y = std::chrono::duration_cast<std::chrono::microseconds>(checkpoint[2] - checkpoint[1]).count();

    cout << "Your single CPU thread is "<<((float)x)/y<<" times faster than your single GPU thread."<< endl;
    return 0;
}

